﻿#include <thrust/device_vector.h>
#include <thrust/swap.h>
#include <thrust/host_vector.h>
#include <thrust/sequence.h>

#include <numeric>

void permutations(thrust::device_vector<char> ch, int l, int r) {
	if (l == r) {
		thrust::copy(ch.begin(), ch.end(), std::ostream_iterator<char>(std::cout));
		std::cout << std::endl;
	}

	for (int i = l; i < r; i++) {
		thrust::swap(*(ch.begin() + l), *(ch.begin() + i));
		permutations(ch, l + 1, r);
		thrust::swap(*(ch.begin() + l), *(ch.begin() + i));
	}
}

int main()
{
	thrust::device_vector<char> s_d(26);
	thrust::sequence(s_d.begin(), s_d.end(), 'a');

	permutations(s_d, 0, s_d.size() - 1);

	return 0;
}
